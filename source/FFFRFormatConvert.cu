#include <cstdint>
#include <hip/hip_runtime.h>

namespace Ffr {
__device__ __forceinline__ float clamp(const float f, const float a, const float b)
{
    return fmaxf(a, fminf(f, b));
}

__device__ __forceinline__ float3 YUVToRGB(const char3 yuv)
{
    // Get YCbCr values
    const float luma = static_cast<float>(yuv.x);
    const float chromaCb = static_cast<float>(yuv.y) - 128.0f;
    const float chromaCr = static_cast<float>(yuv.z) - 128.0f;

    // Convert to RGB using BT601
    return make_float3(clamp(luma + 1.13983 * chromaCr, 0, 255.0f),
        clamp(luma - 0.39465f * chromaCb - 0.58060f * chromaCr, 0, 255.0f),
        clamp(luma + 2.03211f * chromaCb, 0, 255.0f));
}

struct Pixel2
{
    float3 m_pixels[2];
};

__device__ __forceinline__ Pixel2 getNV12ToRGB(
    const uint32_t x, const uint32_t y, const uint8_t* const source[2], const uint32_t sourceStep)
{
    // NV12 is stored as 2 planes: the first plane contains Y the second plane contains U+V interleaved
    // There are 1 U+V sample for every 2x2 Y block
    //  Y1  Y2  Y3  Y4  Y5  Y5
    //  Y7  Y8  Y9  Y10 Y11 Y12
    //  Y13 Y14 Y15 Y16 Y17 Y18
    //  Y19 Y20 Y21 Y22 Y23 Y24
    //
    //  U1 V1 U2 V2 U2 V3
    //  U4 V4 U5 V5 U6 V6
    //
    //  UV1 is used for Y1 Y2 Y7 Y8
    //  UV2 is used for Y3 Y4 Y9 Y10
    //  UV4 is used for Y13 Y14 Y19 Y20
    // etc.
    // Reading a 2x2 Y block requires 2 memory reads as it is split over 2 rows
    //  To try and be a bit more cache friendly Y is processed in 2 pixels (row) at a time instead of 4
    //  This replaces 2 Y loads at a time with 2 UV loads for each 2xY row

    char3 yuvi[2];
    const uint32_t sourceOffset = y * sourceStep + x;
    yuvi[0].x = source[0][sourceOffset];
    yuvi[1].x = source[0][sourceOffset + 1];

    const uint32_t chromaOffset = y >> 1;
    const uint32_t chromaSourceOffset = chromaOffset * sourceStep + x;
    const uint8_t chromaCb = source[1][chromaSourceOffset];
    const uint8_t chromaCr = source[1][chromaSourceOffset + 1];

    // This doesn't perform any chroma interpolation, this feature would need to be added later if needed

    yuvi[0].y = chromaCb;
    yuvi[0].z = chromaCr;
    yuvi[1].y = chromaCb;
    yuvi[1].z = chromaCr;

    Pixel2 rgb;
    rgb.m_pixels[0] = YUVToRGB(yuvi[0]);
    rgb.m_pixels[1] = YUVToRGB(yuvi[1]);

    return rgb;
}

template<typename T>
__device__ __forceinline__ float3 getRGB(const float3 pixel)
{
    // Normalise float values
    return make_float3(pixel.x / 255.0f, pixel.y / 255.0f, pixel.z / 255.0f);
}

template<uint8_t>
__device__ __forceinline__ char3 getRGB(const float3 pixel)
{
    return make_char3(pixel.x, pixel.y, pixel.z);
}

template<typename T>
__global__ void convertNV12ToRGBP(const uint8_t* const source[2], const uint32_t sourceStep, const uint32_t width,
    const uint32_t height, uint8_t* dest[3], const uint32_t destStep)
{
    const uint32_t x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    Pixel2 pixels = getNV12ToRGB(x, y, source, sourceStep);

    const auto pixel1 = getRGB<T>(pixels.m_pixels[0]);
    const auto pixel2 = getRGB<T>(pixels.m_pixels[1]);
    const uint32_t destOffset = y * destStep + x;
    dest[0][destOffset] = pixel1.x;
    dest[0][destOffset + 1] = pixel2.x;
    dest[1][destOffset] = pixel1.y;
    dest[1][destOffset + 1] = pixel2.y;
    dest[2][destOffset] = pixel1.z;
    dest[2][destOffset + 1] = pixel2.z;
}

__forceinline__ int divUp(const uint32_t total, const uint32_t grain)
{
    return (total + grain - 1) / grain;
}

hipError_t convertNV12ToRGB8P(const uint8_t* const source[2], const uint32_t sourceStep, const uint32_t width,
    const uint32_t height, uint8_t* dest[3], const uint32_t destStep)
{
    const dim3 blockDim(8, 8, 1);
    const dim3 gridDim(divUp(width, blockDim.x), divUp(height, blockDim.y), 1);

    convertNV12ToRGBP<char3><<<gridDim, blockDim>>>(source, sourceStep, width, height, dest, destStep);

    return hipPeekAtLastError();
}

hipError_t convertNV12ToRGB32FP(const uint8_t* const source[2], const uint32_t sourceStep, const uint32_t width,
    const uint32_t height, uint8_t* dest[3], const uint32_t destStep)
{
    const dim3 blockDim(8, 8, 1);
    const dim3 gridDim(divUp(width, blockDim.x), divUp(height, blockDim.y), 1);

    convertNV12ToRGBP<char3><<<gridDim, blockDim>>>(source, sourceStep, width, height, dest, destStep);

    return hipPeekAtLastError();
}
} // namespace Ffr