/**
 * Copyright 2019 Matthew Oliver
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <cstdint>

__device__ __forceinline__ float clamp(const float f, const float a, const float b)
{
    return fmaxf(a, fminf(f, b));
}

__device__ __forceinline__ float3 YUVToRGB(const uchar3 yuv)
{
    // Get YCbCr values
    const float luma = static_cast<float>(yuv.x);
    const float chromaCb = static_cast<float>(yuv.y) - 128.0f;
    const float chromaCr = static_cast<float>(yuv.z) - 128.0f;

    // Convert to RGB using BT601
    return make_float3(
        luma + 1.13983 * chromaCr, luma - 0.39465f * chromaCb - 0.58060f * chromaCr, luma + 2.03211f * chromaCb);
}

struct Pixel2
{
    float3 m_pixels[2];
};

struct NV12Planes
{
    uint8_t* m_plane1;
    uint8_t* m_plane2;
};

template<typename T>
struct RGBPlanes
{
    T* m_plane1;
    T* m_plane2;
    T* m_plane3;
};

__device__ __forceinline__ Pixel2 getNV12ToRGB(
    const uint32_t x, const uint32_t y, const NV12Planes source, const uint32_t sourceStep)
{
    // NV12 is stored as 2 planes: the first plane contains Y the second plane contains U+V interleaved
    // There are 1 U+V sample for every 2x2 Y block
    //  Y1  Y2  Y3  Y4  Y5  Y5
    //  Y7  Y8  Y9  Y10 Y11 Y12
    //  Y13 Y14 Y15 Y16 Y17 Y18
    //  Y19 Y20 Y21 Y22 Y23 Y24
    //
    //  U1 V1 U2 V2 U2 V3
    //  U4 V4 U5 V5 U6 V6
    //
    //  UV1 is used for Y1 Y2 Y7 Y8
    //  UV2 is used for Y3 Y4 Y9 Y10
    //  UV4 is used for Y13 Y14 Y19 Y20
    // etc.
    // Reading a 2x2 Y block requires 2 memory reads as it is split over 2 rows
    //  To try and be a bit more cache friendly Y is processed in 2 pixels (row) at a time instead of 4
    //  This replaces 2 Y loads at a time with 2 UV loads for each 2xY row

    uchar3 yuvi[2];
    const uint32_t sourceOffset = y * sourceStep + x;
    yuvi[0].x = source.m_plane1[sourceOffset];
    yuvi[1].x = source.m_plane1[sourceOffset + 1];

    const uint32_t chromaOffset = y >> 1;
    const uint32_t chromaSourceOffset = chromaOffset * sourceStep + x;
    const uint8_t chromaCb = source.m_plane2[chromaSourceOffset];
    const uint8_t chromaCr = source.m_plane2[chromaSourceOffset + 1];

    // This doesn't perform any chroma interpolation, this feature would need to be added later if needed

    yuvi[0].y = chromaCb;
    yuvi[0].z = chromaCr;
    yuvi[1].y = chromaCb;
    yuvi[1].z = chromaCr;

    Pixel2 rgb;
    rgb.m_pixels[0] = YUVToRGB(yuvi[0]);
    rgb.m_pixels[1] = YUVToRGB(yuvi[1]);

    return rgb;
}

template<typename T>
class UpPack
{
public:
    typedef float3 Type;
};

template<>
class UpPack<uint8_t>
{
public:
    typedef uchar3 Type;
};

template<typename T>
__device__ __forceinline__ T getRGB(const float3 pixel)
{
    // Normalise float values
    return make_float3(__saturatef(pixel.x / 255.0f), __saturatef(pixel.y / 255.0f), __saturatef(pixel.z / 255.0f));
}

template<>
__device__ __forceinline__ uchar3 getRGB(const float3 pixel)
{
    return make_uchar3(clamp(pixel.x, 0.0f, 255.0f), clamp(pixel.y, 0.0f, 255.0f), clamp(pixel.z, 0.0f, 255.0f));
}

template<typename T>
__device__ __forceinline__ void convertNV12ToRGBP(const NV12Planes source, const uint32_t sourceStep,
    const uint32_t width, const uint32_t height, RGBPlanes<T> dest, const uint32_t destStep)
{
    const uint32_t x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width - 1 || y >= height) {
        return;
    }

    Pixel2 pixels = getNV12ToRGB(x, y, source, sourceStep);

    const auto pixel1 = getRGB<typename UpPack<T>::Type>(pixels.m_pixels[0]);
    const auto pixel2 = getRGB<typename UpPack<T>::Type>(pixels.m_pixels[1]);
    const uint32_t destOffset = y * destStep + x;
    dest.m_plane1[destOffset] = pixel1.x;
    dest.m_plane1[destOffset + 1] = pixel2.x;
    dest.m_plane2[destOffset] = pixel1.y;
    dest.m_plane2[destOffset + 1] = pixel2.y;
    dest.m_plane3[destOffset] = pixel1.z;
    dest.m_plane3[destOffset + 1] = pixel2.z;
}

extern "C" {
__global__ void convertNV12ToRGB8P(const NV12Planes source, const uint32_t sourceStep, const uint32_t width,
    const uint32_t height, const RGBPlanes<uint8_t> dest, const uint32_t destStep)
{
    convertNV12ToRGBP<uint8_t>(source, sourceStep, width, height, dest, destStep);
}

__global__ void convertNV12ToRGB32FP(const NV12Planes source, const uint32_t sourceStep, const uint32_t width,
    const uint32_t height, const RGBPlanes<float> dest, const uint32_t destStep)
{
    convertNV12ToRGBP<float>(source, sourceStep, width, height, dest, destStep / sizeof(float));
}
}